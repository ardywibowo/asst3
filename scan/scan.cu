#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>
#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256

// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result

template <int TPB>
__global__ void block_exclusive_scan(const int* __restrict__ in, int* __restrict__ out,
                                     int N, int* __restrict__ block_sums) {
    __shared__ int sh[TPB];

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    int x = (gid < N) ? in[gid] : 0;
    sh[tid] = x;
    __syncthreads();

    for (int offset = 1; offset < TPB; offset <<= 1) {
        int t = (tid >= offset) ? sh[tid - offset] : 0;
        __syncthreads();
        if (tid >= offset) sh[tid] += t;
        __syncthreads();
    }

    // Convert to exclusive by shifting right and inserting 0 at start
    int excl = (tid == 0) ? 0 : sh[tid - 1];

    if (gid < N) out[gid] = excl;

    // Write per-block total (the inclusive last element) for carry-out
    if (block_sums && tid == TPB - 1) {
        // Last thread in block holds the block total in sh[TPB-1]
        block_sums[blockIdx.x] = sh[TPB - 1];
    } else if (block_sums && (gid + (TPB - 1 - tid)) >= N && tid == (N - 1 - blockIdx.x * TPB)) {
        // Handle short last block: the last *valid* thread writes the sum
        block_sums[blockIdx.x] = sh[tid];
    }
}

// Kernel 2: uniform add — add each block’s scanned offset to its elements
template <int TPB>
__global__ void add_block_offsets(int* __restrict__ out, int N,
                                  const int* __restrict__ scanned_block_sums) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    if (bid == 0) return;  // first block has zero offset
    int offset = scanned_block_sums[bid - 1];
    if (gid < N) out[gid] += offset;
}

void exclusive_scan(const int* d_in, int N, int* d_out) {
    constexpr int TPB = 256;
    int num_blocks = (N + TPB - 1) / TPB;

    int* d_block_sums = nullptr;
    if (num_blocks > 1) hipMalloc(&d_block_sums, num_blocks * sizeof(int));

    block_exclusive_scan<TPB><<<num_blocks, TPB>>>(d_in, d_out, N, d_block_sums);

    if (num_blocks > 1) {
        if (num_blocks <= TPB) {
            block_exclusive_scan<TPB><<<1, TPB>>>(d_block_sums, d_block_sums, num_blocks, nullptr);
        } else {
            block_exclusive_scan<TPB><<<(num_blocks + TPB - 1) / TPB, TPB>>>(d_block_sums, d_block_sums, num_blocks, nullptr);
        }

        // 3) Uniform add per block
        add_block_offsets<TPB><<<num_blocks, TPB>>>(d_out, N, d_block_sums);
        hipFree(d_block_sums);
    }
}

//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray) {
    int* device_result;
    int* device_input;
    int N = end - inarray;

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);

    hipMalloc((void**)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void**)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration;
}

// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {
    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration;
}

// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {
    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.

    return 0;
}

//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int* input, int length, int* output, int* output_length) {
    int* device_input;
    int* device_output;
    int rounded_length = nextPow2(length);

    hipMalloc((void**)&device_input, rounded_length * sizeof(int));
    hipMalloc((void**)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();

    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime;
    return duration;
}

void printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
